#include "utils.cuh"

bool initialized = false;

hipStream_t calc_stream;
hipblasHandle_t cublas_handle;

int graphCreated_padding_length = -1;
int graphCreated_input_length = -1;
hipGraph_t graph;
hipGraphExec_t graphExec;

void init_resources() {
  if (initialized) return;
  cudaCheck(hipStreamCreate(&calc_stream));
  cublasCheck(hipblasCreate(&cublas_handle));
  cublasCheck(hipblasSetStream(cublas_handle, calc_stream));
  initialized = true;
}