#include "utils.cuh"

bool initialized = false;

hipStream_t calc_stream;
hipblasHandle_t cublas_handle;

bool graphCreated = false;
hipGraph_t graph;
hipGraphExec_t graphExec;

void init_resources() {
  if (initialized) return;
  cudaCheck(hipStreamCreate(&calc_stream));
  cublasCheck(hipblasCreate(&cublas_handle));
  cublasCheck(hipblasSetStream(cublas_handle, calc_stream));
  initialized = true;
}