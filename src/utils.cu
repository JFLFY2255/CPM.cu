#include "utils.cuh"

bool initialized = false;
hipblasHandle_t cublas_handle;

void init_cublas() {
  if (initialized) return;
  cublasCheck(hipblasCreate(&cublas_handle));
  initialized = true;
}